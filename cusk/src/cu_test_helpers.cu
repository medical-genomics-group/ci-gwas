#include "hip/hip_runtime.h"
#include <mps/cu_test_helpers.h>
#include <mps/gpuerrors.h>

void test_cal_Indepl0(
    const float *C, const int *M, const int *P, const int *W, int *G, const float *Th
)
{
    float *C_cuda;
    int *G_cuda;
    float *pMax_cuda;

    // num phen
    int p = *P;
    // num markers
    int m = *M;
    // corr width
    int w = *W;
    // height (nrows) of the corr matrix: num_markers + num_phen
    int nr = p + m;
    // width (ncols) of the corr matrix: corr_width + num_phen
    int nc = w + p;

    int max_marker_degree = 2 * w + p;
    int max_phen_degree = m + p;
    int mixed_matrix_size = max_marker_degree * m + max_phen_degree * p;

    HANDLE_ERROR(hipMalloc((void **)&C_cuda, nc * nr * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void **)&G_cuda, mixed_matrix_size * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void **)&pMax_cuda, mixed_matrix_size * sizeof(float)));
    // copy correlation matrix from CPU to GPU
    HANDLE_ERROR(hipMemcpy(C_cuda, C, nc * nr * sizeof(float), hipMemcpyHostToDevice));
    CudaCheckError();

    dim3 BLOCKS_PER_GRID;
    dim3 THREADS_PER_BLOCK;

    if ((nc * nr) < 1024)
    {
        BLOCKS_PER_GRID = dim3(1, 1, 1);
        THREADS_PER_BLOCK = dim3(nr, nc, 1);
        cal_Indepl0<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(
            C_cuda, G_cuda, Th[0], pMax_cuda, m, p, w
        );
        CudaCheckError();
    }
    else
    {
        BLOCKS_PER_GRID = dim3(ceil(((float)nr) / 32.0), ceil(((float)nc) / 32.0), 1);
        THREADS_PER_BLOCK = dim3(32, 32, 1);
        cal_Indepl0<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(
            C_cuda, G_cuda, Th[0], pMax_cuda, m, p, w
        );
        CudaCheckError();
    }

    // Copy Graph G from GPU to CPU
    HANDLE_ERROR(hipMemcpy(G, G_cuda, mixed_matrix_size * sizeof(int), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipFree(C_cuda));
    HANDLE_ERROR(hipFree(G_cuda));
    HANDLE_ERROR(hipFree(pMax_cuda));
    CudaCheckError();
}

// void test_cal_scan_compact(int *G_compact, const int *G, int *nprime, const int *M, const int *P,
// const int *W)
// {
//     int *G_cuda;
//     int *G_compact_cuda;
//     int *nprime_cuda;

//     int nr = w + p;
//     int max_marker_degree = 2 * w + p;
//     int max_phen_degree = nr;
//     int adj_mat_size = max_marker_degree * m + max_phen_degree * p;
//     int compact_adj_mat_size = max_marker_degree * m + max_phen_degree * p + nr;

//     // num phen
//     int p = *P;
//     // num markers
//     int m = *M;
//     // corr width
//     int w = *W;

//     HANDLE_ERROR(hipMalloc((void **)&G_cuda, adj_mat_size * sizeof(int)));
//     HANDLE_ERROR(hipMalloc((void **)&G_compact_cuda, compact_adj_mat_size * sizeof(int)));
//     HANDLE_ERROR(hipMalloc((void **)&nprime_cuda, 1 * sizeof(int)));
//     HANDLE_ERROR(hipMemcpy(G_cuda, G, adj_mat_size * sizeof(int), hipMemcpyHostToDevice));
//     HANDLE_ERROR(hipMemset(nprime_cuda, 0, 1 * sizeof(int)));
//     CudaCheckError();

//     BLOCKS_PER_GRID = dim3(1, nr, 1);
//     THREADS_PER_BLOCK = dim3(1024, 1, 1);

//     scan_compact<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK, nr * sizeof(int)>>>(
//         GPrime_cuda, G_cuda, nr, nprime_cuda);
//     CudaCheckError();

//     // Copy results back
//     HANDLE_ERROR(hipMemcpy(nprime, nprime_cuda, 1 * sizeof(int), hipMemcpyDeviceToHost));
//     HandleError(hipMemcpy(G_compact, G_compact_cuda, compact_adj_mat_size * sizeof(int),
//     hipMemcpyDeviceToHost)); CudaCheckError();
// }