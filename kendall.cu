#include "hip/hip_runtime.h"
#include <math.h>

#include "bed_lut.h"
#include "gpuerrors.h"
#include "kendall.h"

// Compute correlation matrix of all pairs of markers in 'x'.
// The data is not partitioned.
// 'x' is assumed to hold uncompressed genomic marker values without NaN.
void cu_corr_npn(const unsigned char *marker_vals, const size_t num_markers,
                 const size_t num_individuals, float *results)
{
    // This here assumes a non compressed a.
    size_t marker_vals_bytes = num_markers * num_individuals * sizeof(unsigned char);
    unsigned char *gpu_marker_vals;
    float *gpu_results;
    int threads_per_block = NUMTHREADS;
    size_t output_length = num_markers * (num_markers - 1) / 2;
    size_t output_bytes = output_length * sizeof(float);

    // TODO: see if proper blocks give any performace increase
    int blocks_per_grid = output_length;

    HANDLE_ERROR(hipMalloc(&gpu_marker_vals, marker_vals_bytes));
    HANDLE_ERROR(
        hipMemcpy(gpu_marker_vals, marker_vals, marker_vals_bytes, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMalloc(&gpu_results, output_bytes));

    cu_marker_corr_npn<<<blocks_per_grid, threads_per_block>>>(gpu_marker_vals, num_markers,
                                                               num_individuals, gpu_results);

    HANDLE_ERROR(hipMemcpy(results, gpu_results, output_bytes, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipFree(gpu_marker_vals));
    HANDLE_ERROR(hipFree(gpu_results));
}

// A O(n) runtime Kendall implementation for uncompressed genomic marker data.
__global__ void cu_marker_corr_npn(const unsigned char *marker_vals, const size_t num_markers,
                                   const size_t num_individuals, float *results)
{
    size_t tix = threadIdx.x;

    // convert linear indices into correlation matrix into (row, col) ix
    size_t lin_ix = blockIdx.x;
    float lin_ix_f = (size_t)lin_ix;
    float l = num_markers - 1;
    float b = 2 * l - 1;
    float c = 2 * (l - lin_ix_f);
    float row = std::floor((-b + sqrt(b * b + 4 * c)) / -2.0) + 1.0;
    float h = (-(row * row) + row * (2.0 * l + 1.0)) / 2.0;
    // offset of 1 because we don't compute the diagonal
    float col = lin_ix_f - h + row + 1;
    size_t col_start_a = row * num_individuals;
    size_t col_start_b = col * num_individuals;

    float thread_sum[9] = {0.0};
    __shared__ float thread_sums[NUMTHREADS][9];
    // TODO: it seems stupid to jump in memory, sequential reads are probably more efficient.
    // should have ++ increment and adjust the start.
    for (size_t i = tix; i < num_individuals; i += NUMTHREADS) {
        thread_sum[(3 * marker_vals[col_start_a + i]) + marker_vals[col_start_b + i]] += 1.f;
    }

    for (size_t i = 0; i < 9; i++) {
        thread_sums[tix][i] = thread_sum[i];
    }

    // consolidate thread_sums
    __syncthreads();
    if (tix == 0) {
        // printf("block [x: %f; y: %f]: making single sum", row, col);

        // produce single sum
        float s[9] = {0.0};
        for (size_t i = 0; i < NUMTHREADS; i++) {
            for (size_t j = 0; j < 9; j++) {
                s[j] += thread_sums[i][j];
            }
        }
        float p = ((s[0] * (s[4] + s[5] + s[7] + s[8])) + (s[1] * (s[5] + s[8])) +
                   (s[3] * (s[7] + s[8])) + (s[4] * s[8]));
        float q = ((s[1] * (s[3] + s[6])) + (s[2] * (s[3] + s[4] + s[6] + s[7])) + (s[4] * s[6]) +
                   (s[5] * (s[6] + s[7])));
        float t = ((s[0] * (s[1] + s[2])) + (s[1] * s[2]) + (s[3] * (s[4] + s[5])) + (s[4] * s[5]) +
                   (s[6] * (s[7] + s[8])) + (s[7] * s[8]));
        float u = ((s[0] * (s[3] + s[6])) + (s[1] * (s[4] + s[7])) + (s[2] * (s[5] + s[8])) +
                   (s[3] * s[6]) + (s[4] * s[7]) + (s[5] * s[8]));

        float kendall_corr = (p - q) / sqrt((p + q + t) * (p + q + u));

        // printf("linear ix: %f, row: %f, col: %f, h: %f, l: %f, corr result: %f \n", lin_ix_f,
        // row,
        //        col, h, l, kendall_corr);

        results[lin_ix] = sin(M_PI / 2 * kendall_corr);
    }
}

// Kendall correlation computation for pairs of markers compressed in .bed format
// without leading magic numbers.
// Correlation matrix is computed in one go, without any partitioning.
void cu_corr_npn(const unsigned char *marker_vals, const float *phen_vals, const size_t num_markers,
                 const size_t num_individuals, const size_t num_phen, const float marker_mean,
                 const float marker_std, float *results)
{
    // this is ceil
    size_t col_len_bytes = (num_individuals + 3) / 4 * sizeof(unsigned char);
    size_t marker_vals_bytes = col_len_bytes * num_markers;
    unsigned char *gpu_marker_vals;
    float *gpu_results;
    int threads_per_block = NUMTHREADS;
    size_t output_length = num_markers * (num_markers - 1) / 2;
    size_t output_bytes = output_length * sizeof(float);

    // TODO: see if proper blocks give any performace increase
    int blocks_per_grid = output_length;

    HANDLE_ERROR(hipMalloc(&gpu_marker_vals, marker_vals_bytes));
    HANDLE_ERROR(
        hipMemcpy(gpu_marker_vals, marker_vals, marker_vals_bytes, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMalloc(&gpu_results, output_bytes));

    bed_marker_corr_npn<<<blocks_per_grid, threads_per_block>>>(
        gpu_marker_vals, num_markers, num_individuals, col_len_bytes, gpu_results);
    CudaCheckError();

    HANDLE_ERROR(hipMemcpy(results, gpu_results, output_bytes, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipFree(gpu_marker_vals));
    HANDLE_ERROR(hipFree(gpu_results));
}

__global__ void marker_phen_corr_pearson(const unsigned char *marker_vals, const float *phen_vals,
                                         const size_t num_markers, const size_t num_individuals,
                                         const size_t num_phen, const size_t col_len_bytes,
                                         const float marker_mean, const float marker_std,
                                         float *results)
{
    size_t tix = threadIdx.x;
    size_t lin_ix = blockIdx.x;
    size_t mv_ix = lin_ix / num_phen;
    size_t phen_ix = lin_ix - (num_phen * mv_ix);
    size_t mv_start_ix = mv_ix * col_len_bytes;
    size_t phen_start_ix = phen_ix * num_individuals;

    float thread_sum_mv_phen = 0.0;
    float thread_sum_phen = 0.0;
    __shared__ float thread_sums_mv_phen[NUMTHREADS];
    __shared__ float thread_sums_phen[NUMTHREADS];
    for (size_t i = tix; i < col_len_bytes; i += NUMTHREADS) {
        size_t curr_mv_byte_ix = 4 * (size_t)(marker_vals[mv_start_ix + i]);
        for (size_t j = 0; (j < 4) && (i * 4 + j < num_individuals); j++) {
            float mv_val = bed_lut_a[(curr_mv_byte_ix + j)];
            float phen_val = phen_vals[(phen_start_ix + i + j)];
            thread_sum_mv_phen += mv_val * phen_val;
            thread_sum_phen += phen_val;
        }
    }

    thread_sums_mv_phen[tix] = thread_sum_mv_phen;
    thread_sums_phen[tix] = thread_sum_phen;

    __syncthreads();
    if (tix = 0) {
        float s_mv_phen = 0.0;
        float s_phen = 0.0;
        for (size_t i = 0; i < NUMTHREADS; i++) {
            s_mv_phen += thread_sums_mv_phen[i];
            s_phen += thread_sums_phen[i];
        }

        results[lin_ix] =
            (s_mv_phen - marker_mean * s_phen) / ((float)(num_individuals - 1) * marker_std);
    }
}

// Compute Pearson's r between pairs of standardized phenotype vectors.
__global__ void phen_corr_pearson(const float *phen_vals, const size_t num_individuals,
                                  const size_t num_phen, float *results)
{
    size_t tix = threadIdx.x;

    // convert linear indices into correlation matrix into (row, col) ix
    size_t lin_ix = blockIdx.x;
    float lin_ix_f = (size_t)lin_ix;
    float l = num_phen - 1;
    float b = 2 * l - 1;
    float c = 2 * (l - lin_ix_f);
    float row = std::floor((-b + sqrt(b * b + 4 * c)) / -2.0) + 1.0;
    float h = (-(row * row) + row * (2.0 * l + 1.0)) / 2.0;
    // offset of 1 because we don't compute the diagonal
    float col = lin_ix_f - h + row + 1;
    size_t col_start_a = row * num_individuals;
    size_t col_start_b = col * num_individuals;

    float thread_sum = 0.0;
    __shared__ float thread_sums[NUMTHREADS];
    for (size_t i = tix; i < num_individuals; i += NUMTHREADS) {
        float val_a = phen_vals[(col_start_a + i)];
        float val_b = phen_vals[(col_start_b + i)];
        thread_sum += val_a * val_b;
    }

    thread_sums[tix] = thread_sum;

    __syncthreads();
    if (tix = 0) {
        float s = 0.0 for (size_t i = 0; i < NUMTHREADS; i++) { s += thread_sums[i]; }

        results[lin_ix] = s / (float)(num_individuals - 1);
    }
}

// A O(n) runtime Kendall implementation for compressed genomic marker data.
// The compression format is expected to be col-major .bed without NaN
// and without leading magic numbers.
__global__ void bed_marker_corr_npn(const unsigned char *marker_vals, const size_t num_markers,
                                    const size_t num_individuals, const size_t col_len_bytes,
                                    float *results)
{
    size_t tix = threadIdx.x;

    // convert linear indices into correlation matrix into (row, col) ix
    size_t lin_ix = blockIdx.x;
    float lin_ix_f = (size_t)lin_ix;
    float l = num_markers - 1;
    float b = 2 * l - 1;
    float c = 2 * (l - lin_ix_f);
    float row = std::floor((-b + sqrt(b * b + 4 * c)) / -2.0) + 1.0;
    float h = (-(row * row) + row * (2.0 * l + 1.0)) / 2.0;
    // offset of 1 because we don't compute the diagonal
    float col = lin_ix_f - h + row + 1;
    size_t col_start_a = row * col_len_bytes;
    size_t col_start_b = col * col_len_bytes;

    float thread_sum[9] = {0.0};
    __shared__ float thread_sums[NUMTHREADS][9];

    // TODO: it seems stupid to jump in memory, sequential reads are probably more efficient.
    // should have ++ increment and adjust the start.
    for (size_t i = tix; i < col_len_bytes; i += NUMTHREADS) {
        size_t aix = 4 * (size_t)(marker_vals[col_start_a + i]);
        size_t bix = 4 * (size_t)(marker_vals[col_start_b + i]);
        for (size_t j = 0; (j < 4) && (i * 4 + j < num_individuals); j++) {
            float val_a = bed_lut_a[(aix + j)];
            float val_b = bed_lut_a[(bix + j)];
            size_t comp_ix = (size_t)((3 * val_a + val_b));
            thread_sum[comp_ix] += 1.f;
        }
    }

    for (size_t i = 0; i < 9; i++) {
        thread_sums[tix][i] = thread_sum[i];
    }

    // consolidate thread_sums
    __syncthreads();
    if (tix == 0) {
        // produce single sum
        float s[9] = {0.0};
        for (size_t i = 0; i < NUMTHREADS; i++) {
            for (size_t j = 0; j < 9; j++) {
                s[j] += thread_sums[i][j];
            }
        }
        float p = ((s[0] * (s[4] + s[5] + s[7] + s[8])) + (s[1] * (s[5] + s[8])) +
                   (s[3] * (s[7] + s[8])) + (s[4] * s[8]));
        float q = ((s[1] * (s[3] + s[6])) + (s[2] * (s[3] + s[4] + s[6] + s[7])) + (s[4] * s[6]) +
                   (s[5] * (s[6] + s[7])));
        float t = ((s[0] * (s[1] + s[2])) + (s[1] * s[2]) + (s[3] * (s[4] + s[5])) + (s[4] * s[5]) +
                   (s[6] * (s[7] + s[8])) + (s[7] * s[8]));
        float u = ((s[0] * (s[3] + s[6])) + (s[1] * (s[4] + s[7])) + (s[2] * (s[5] + s[8])) +
                   (s[3] * s[6]) + (s[4] * s[7]) + (s[5] * s[8]));

        float kendall_corr = (p - q) / sqrt((p + q + t) * (p + q + u));

        results[lin_ix] = sin(M_PI / 2 * kendall_corr);
    }
}
